/**
 * Museqa: Multiple Sequence Aligner using hybrid parallel computing.
 * @file CUDA wrapper global variables and functions definitions.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2018-present Rodrigo Siqueira
 */
#if !defined(MUSEQA_AVOID_CUDA)

#include <hip/hip_runtime.h>

#include <string>
#include <cstdint>

#include <museqa/utility.hpp>

#include <museqa/cuda/common.cuh>
#include <museqa/cuda/device.cuh>
#include <museqa/cuda/stream.cuh>
#include <museqa/cuda/event.cuh>

namespace museqa
{
    using namespace cuda;

    /**
     * Retrieves a brief textual explanation for a specified kind of CUDA runtime
     * API status or error code.
     * @param code The error code to be described.
     * @return The error description.
     */
    auto cuda::error::describe(error::code code) noexcept -> std::string
    {
        return hipGetErrorString(code);
    }

    /**
     * Retrieves the total amount of memory available for immediate usage within
     * the currently active device.
     * @return The amount of memory available in the current device.
     */
    auto cuda::device::memory::available() noexcept(!safe) -> size_t
    {
        size_t available, total;
        cuda::check(hipMemGetInfo(&available, &total));
        return available;
    }

    /**
     * Retrieves the total amount of memory available for immediate usage within
     * the given target device.
     * @param target The device to be introspected.
     * @return The amount of memory available in the selected device.
     */
    auto cuda::device::memory::available(device::id target) noexcept(!safe) -> size_t
    {
        cuda::device::current::scope temporary {target};
        return cuda::device::memory::available();
    }

    /**
     * Retrieves the total amount of global memory present in the currently active
     * device's hardware, independently whether this memory is available or not.
     * @return The total amount of global memory within the current device.
     */
    auto cuda::device::memory::total() noexcept(!safe) -> size_t
    {
        size_t available, total;
        cuda::check(hipMemGetInfo(&available, &total));
        return total;
    }

    /**
     * Retrieves the total amount of global memory present in the given device's
     * hardware, independently whether this memory is available or not.
     * @param target The device to be introspected.
     * @return The total amount of global memory within the selected device.
     */
    auto cuda::device::memory::total(device::id target) noexcept(!safe) -> size_t
    {
        cuda::device::current::scope temporary {target};
        return cuda::device::memory::total();
    }

    /**
     * Retrieves the currently active device.
     * @return The current active device.
     */
    __host__ __device__ auto cuda::device::current::get() noexcept(!safe) -> cuda::device::id
    {
        cuda::device::id device;
        cuda::check(hipGetDevice(&device));
        return device;
    }

    /**
     * Changes the currently active device to the given one.
     * @param target The new device to be active.
     */
    void cuda::device::current::set(cuda::device::id target) noexcept(!safe)
    {
        cuda::check(hipSetDevice(target));
    }

    /**
     * Changes the currently active device and returns the one previously active.
     * @param target The new device to be active.
     * @return The previously active device.
     */
    auto cuda::device::current::scope::replace(cuda::device::id target) noexcept(!safe) -> cuda::device::id
    {
        auto previous = cuda::device::current::get();
        cuda::device::current::set(target);
        return previous;
    }

    /**
     * Retrieves the total number of compute-capable devices currently available
     * and directly accessible to the current process.
     * @return The total number of directly accessible devices.
     */
    __host__ __device__ auto cuda::device::count() noexcept(!safe) -> size_t
    {
        int total = 0;
        cuda::check(hipGetDeviceCount(&total));
        return static_cast<size_t>(total);
    }

    /**
     * Checks whether the given stream has finished executing its queue.
     * @return Has the stream completed all its tasks?
     */
    bool cuda::stream::ready() const noexcept(!safe)
    {
        return cuda::ready(hipStreamQuery(*this));
    }

    /**
     * Blocks execution and waits for all stream tasks to complete.
     * @see museqa::cuda::synchronize
     */
    void cuda::stream::synchronize() const noexcept(!safe)
    {
        cuda::device::current::scope temporary {m_device};
        cuda::check(hipStreamSynchronize(*this));
    }

    /**
     * Blocks stream execution and waits for the given event to be fired.
     * The event does not need to be on the same device as the stream, thus
     * allowing synchronization between different devices.
     * @param event The event to waited on.
     * @see museqa::cuda::event
     */
    void cuda::stream::wait(cuda::event::id event) const noexcept(!safe)
    {
        cuda::check(hipStreamWaitEvent(*this, event, 0u));
    }

    /**
     * Checks whether event's recorded stream has completed its the work.
     * @return Has all recorded work been completed?
     */
    bool cuda::event::ready() const noexcept(!safe)
    {
        return cuda::ready(hipEventQuery(*this));
    }

    /**
     * Captures the contents of a stream at the time of this call.
     * @param stream The stream to have its contents captured by the event.
     * @note Both the event and the stream must be in the same device.
     */
    void cuda::event::record(cuda::stream::id stream) noexcept(!safe)
    {
        cuda::check(hipEventRecord(*this, stream));
    }

    /**
     * Waits until the completion of all work currently captured by event.
     * @see museqa::cuda::synchronize
     */
    void cuda::event::synchronize() const noexcept(!safe)
    {
        cuda::device::current::scope temporary {m_device};
        cuda::check(hipEventSynchronize(*this));
    }
}

#endif

/**
 * Museqa: Multiple Sequence Aligner using hybrid parallel computing.
 * @file CUDA wrapper global variables and functions definitions.
 * @author Rodrigo Siqueira <rodriados@gmail.com>
 * @copyright 2018-present Rodrigo Siqueira
 */
#if !defined(MUSEQA_AVOID_CUDA)

#include <hip/hip_runtime.h>

#include <string>
#include <cstdint>

#include <museqa/utility.hpp>

#include <museqa/cuda/common.cuh>
#include <museqa/cuda/device.cuh>
#include <museqa/cuda/stream.cuh>
#include <museqa/cuda/event.cuh>

namespace museqa
{
    using namespace cuda;

    /**
     * Retrieves a brief textual explanation for a specified kind of CUDA runtime
     * API status or error code.
     * @param code The error code to be described.
     * @return The error description.
     */
    auto cuda::error::describe(error::code code) noexcept -> std::string
    {
        return hipGetErrorString(code);
    }

    /**
     * Retrieves the total amount of memory available for immediate usage within
     * the currently active device.
     * @return The amount of memory available in the current device.
     */
    auto cuda::device::memory::available() noexcept(!safe) -> size_t
    {
        size_t available, total;
        cuda::check(hipMemGetInfo(&available, &total));
        return available;
    }

    /**
     * Retrieves the total amount of memory available for immediate usage within
     * the given target device.
     * @param target The device to be introspected.
     * @return The amount of memory available in the selected device.
     */
    auto cuda::device::memory::available(device::id target) noexcept(!safe) -> size_t
    {
        cuda::device::current::scope temporary {target};
        return cuda::device::memory::available();
    }

    /**
     * Retrieves the total amount of global memory present in the currently active
     * device's hardware, independently whether this memory is available or not.
     * @return The total amount of global memory within the current device.
     */
    auto cuda::device::memory::total() noexcept(!safe) -> size_t
    {
        size_t available, total;
        cuda::check(hipMemGetInfo(&available, &total));
        return total;
    }

    /**
     * Retrieves the total amount of global memory present in the given device's
     * hardware, independently whether this memory is available or not.
     * @param target The device to be introspected.
     * @return The total amount of global memory within the selected device.
     */
    auto cuda::device::memory::total(device::id target) noexcept(!safe) -> size_t
    {
        cuda::device::current::scope temporary {target};
        return cuda::device::memory::total();
    }

    /**
     * Retrieves the currently active device.
     * @return The current active device.
     */
    __host__ __device__ auto cuda::device::current::get() noexcept(!safe) -> cuda::device::id
    {
        cuda::device::id device;
        cuda::check(hipGetDevice(&device));
        return device;
    }

    /**
     * Changes the currently active device to the given one.
     * @param target The new device to be active.
     */
    void cuda::device::current::set(cuda::device::id target) noexcept(!safe)
    {
        cuda::check(hipSetDevice(target));
    }

    /**
     * Changes the currently active device and returns the one previously active.
     * @param target The new device to be active.
     * @return The previously active device.
     */
    auto cuda::device::current::scope::replace(cuda::device::id target) noexcept(!safe) -> cuda::device::id
    {
        auto previous = cuda::device::current::get();
        cuda::device::current::set(target);
        return previous;
    }

    /**
     * Retrieves the total number of compute-capable devices currently available
     * and directly accessible to the current process.
     * @return The total number of directly accessible devices.
     */
    __host__ __device__ auto cuda::device::count() noexcept(!safe) -> size_t
    {
        int total = 0;
        cuda::check(hipGetDeviceCount(&total));
        return static_cast<size_t>(total);
    }

    /**
     * Checks whether the given stream has finished executing its queue.
     * @param stream The stream to check completion of.
     * @return Has the stream completed all its tasks?
     */
    bool cuda::stream::ready(const cuda::stream& stream) noexcept(!safe)
    {
        return cuda::ready(hipStreamQuery(stream));
    }

    /**
     * Blocks execution and waits for all stream tasks to complete.
     * @param stream The stream to be synchronized.
     * @see museqa::cuda::synchronize
     */
    void cuda::stream::synchronize(const cuda::stream& stream) noexcept(!safe)
    {
        cuda::device::current::scope temporary {stream.m_device};
        cuda::check(hipStreamSynchronize(stream));
    }

    /**
     * Blocks stream execution and waits for the given event to be fired.
     * The event does not need to be on the same device as the stream, thus
     * allowing synchronization between different devices.
     * @param stream The stream to wait on given the event.
     * @param event The event to waited on.
     * @see museqa::cuda::event
     */
    void cuda::stream::wait(const cuda::stream& stream, cuda::event::id event) noexcept(!safe)
    {
        cuda::check(hipStreamWaitEvent(stream, event, 0u));
    }

    /**
     * Checks whether event's recorded stream has completed its the work.
     * @param event The event to check completion of.
     * @return Has all recorded work been completed?
     */
    bool cuda::event::ready(const cuda::event& event) noexcept(!safe)
    {
        return cuda::ready(hipEventQuery(event));
    }

    /**
     * Captures the contents of a stream at the time of this call.
     * @param event The target event to capture the given stream.
     * @param stream The stream to have its contents captured by the event.
     * @note Both the event and the stream must be in the same device.
     */
    void cuda::event::record(cuda::event& event, cuda::stream::id stream) noexcept(!safe)
    {
        cuda::check(hipEventRecord(event, stream));
    }

    /**
     * Waits until the completion of all work currently captured by event.
     * @param event The event to be synchronized.
     * @see museqa::cuda::synchronize
     */
    void cuda::event::synchronize(const cuda::event& event) noexcept(!safe)
    {
        cuda::device::current::scope temporary {event.m_device};
        cuda::check(hipEventSynchronize(event));
    }
}

#endif
